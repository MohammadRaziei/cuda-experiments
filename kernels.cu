#include "hip/hip_runtime.h"
#include <chrono>
#include <iostream>

#include <ppl.h>
#include "hip/hip_runtime.h"
#include ""
#include "io.h"
#include "thrust/sort.h"

#ifdef __INTELLISENSE__
#define __syncthreads()
#define CUDA_KERNEL(...)
#else
#define CUDA_KERNEL(...) <<< __VA_ARGS__ >>>
#endif

#define MaxMedianWindowSize (300)

/// function to sort the array in ascending order
template <typename T>
__device__ __forceinline__ void arraySort(T* array, const uint32_t n) {
  /// declare some local variables
  T temp = 0;
  for (uint32_t i = 0; i < n; i++) {
    for (uint32_t j = 0; j < n - 1; j++) {
      if (array[j] > array[j + 1]) {
        temp = array[j];
        array[j] = array[j + 1];
        array[j + 1] = temp;
      }
    }
  }
}

// KERNEL: Median Filter
__device__ void sortedInOut(float sortedData[], const uint32_t len,
                            const float& outValue, const float& inValue) {
  bool notFound = true;
  float value = sortedData[0], saveValue;
  sortedData[len] = inValue;
  for (uint32_t j = 0, i = 0; i < len; ++i) {
    if (value == outValue) {
      value = sortedData[++j];
    }
    if (notFound && value >= inValue) {
      sortedData[i] = inValue;
      notFound = false;
    } else {
      saveValue = value;
      value = sortedData[++j];
      sortedData[i] = saveValue;
    }
  }
}
__device__ void medianFilterCore(float output[], const float input[],
                                 const uint32_t len,
                                 const uint32_t halfWindow) {
  const uint32_t windowSize = 2 * halfWindow + 1;
  float temp[MaxMedianWindowSize];
  {
    float* temp_p = temp;
    const float* input_p = input;
    for (uint32_t i = 0; i < windowSize; ++i) *(temp_p++) = *(input_p++);
  }
  arraySort(temp, windowSize);
  output[0] = temp[halfWindow];
  for (uint32_t i = 0; i < len - 1; ++i) {
    sortedInOut(temp, windowSize, input[i], input[i + windowSize]);
    output[i + 1] = temp[halfWindow];
  }
}
__global__ void medianFilterKernel(float output[], const float input[],
                                   const uint32_t len,
                                   const uint32_t halfWindow) {
  const uint32_t numFrames = blockDim.x * gridDim.x;
  const uint32_t lenFrames = (uint32_t)ceilf((float)len / numFrames);
  const uint32_t i = (threadIdx.x + blockDim.x * blockIdx.x) * lenFrames;
  if (i < len) {
    if (i + lenFrames <= len)
      medianFilterCore(output + i, input + i, lenFrames, halfWindow);
    else
      medianFilterCore(output + i, input + i, len - i, halfWindow);
  }
}

// KERNEL: Abs
__global__ void absKernel(float* out, float* real, float* imag, uint32_t size) {
  for (uint32_t i{threadIdx.x + blockIdx.x * blockDim.x}; i < size;
       i += blockDim.x * gridDim.x)
    out[i] = hypotf(real[i], imag[i]);
}

// KERNEL: Subtract
__global__ void mysubtractKernel(float out[], const float in1[],
                                 const float in2[], const uint32_t len) {
  for (uint32_t i = threadIdx.x + blockDim.x * blockIdx.x; i < len;
       i += blockDim.x * gridDim.x) {
    out[i] = in1[i] - in2[i];
  }
}

// KERNEL: sqrHypot
__global__ void sqrHypotKernel(float out[], const float sigReal[],
                               const float sigImag[], const uint32_t len) {
  for (uint32_t i = threadIdx.x + blockDim.x * blockIdx.x; i < len;
       i += blockDim.x * gridDim.x) {
    out[i] = sigReal[i] * sigReal[i] + sigImag[i] * sigImag[i];
  }
}
__global__ void sqrHypotKernel(float out[], const hipComplex sigComp[],
                               const uint32_t len) {
  for (uint32_t i = threadIdx.x + blockDim.x * blockIdx.x; i < len;
       i += blockDim.x * gridDim.x) {
    out[i] = sigComp[i].x * sigComp[i].x + sigComp[i].y * sigComp[i].y;
  }
}
__global__ void sqrHypotKernel(hipComplex out[], const hipComplex sigComp[],
                               const uint32_t len) {
  for (uint32_t i = threadIdx.x + blockDim.x * blockIdx.x; i < len;
       i += blockDim.x * gridDim.x) {
    out[i].x = sigComp[i].x * sigComp[i].x + sigComp[i].y * sigComp[i].y;
  }
}

// KERNEL: sum
__global__ void sumSerialkernel(float* out, const float input[],
                                const uint32_t len, float scale = 1.f) {
  if (0 == (threadIdx.x + blockDim.x * blockIdx.x)) {
    float sumVal = 0;
    for (uint32_t i = 0; i < len; ++i) {
      sumVal += input[i];
    }
    *out = sumVal / scale;
  }
}